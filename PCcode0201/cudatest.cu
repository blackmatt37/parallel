#include <iostream>
#include <hip/hip_runtime.h>

#include <stdio.h>


int WIDTH = 400;
int HEIGHT = 300;
__device__ double translatex(int x)
{
	return x/100.0 - 2.0;

}
__device__ double translatey(int y)
{
	return y/100.0 - 1.5;

}

__global__ void mandel(int* gpu_t)
{
	int tmax = 100;
	double x = translatex(blockIdx.x);
	double y = translatey(blockIdx.y);
	double a = 0.0;
	double b = 0.0;
	int i;
	for(i=0; i<tmax; i++)
	{
		double anew = a*a - b*b + x;
		double bnew = 2*a*b + y;
		a = anew;
		b = bnew;
		if(a*a + b*b > 4.0)
			break;
	}
	*(gpu_t + 400*blockIdx.x + blockIdx.y) = i+1;
}



int main(int argc, char const *argv[])
{
	int *gpu_t;
	hipMalloc((void**)&gpu_t, HEIGHT*WIDTH*sizeof(int));
	int cpu_t[WIDTH][HEIGHT];
	dim3 grid(HEIGHT, WIDTH);
	mandel<<<grid, 1>>>(gpu_t);
	hipMemcpy(cpu_t, gpu_t, HEIGHT*WIDTH*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpu_t);
	int x;
	int y;
	for(x = 0; x<WIDTH; x++)
	{
		for(y = 0;y< HEIGHT; y++)
		{
			printf("%d\n", cpu_t[x][y]);
		}
	}

}
