#include "hip/hip_runtime.h"
#include <stdio.h>
#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int HEIGHT = 300;
int WIDTH = 400;

int tmax = 100;



// int mandel(double, double, int);
void translate(int, int, double*, double*);
__device__ double translatex(int x)
{
	return x*1.0/100.0 - 2.0;

}
__device__ double translatey(int y)
{
	return y*1.0/100.0 - 1.5;

}

__global__ void mandel(int* gpu_t)
{
	int tmax = 100;
	double x = translatex(blockIdx.x);
	double y = translatey(blockIdx.y);
	double a = 0.0;
	double b = 0.0;
	int i;
	for(i=0; i<tmax; i++)
	{
		double anew = a*a - b*b + x;
		double bnew = 2*a*b + y;
		a = anew;
		b = bnew;
		if(a*a + b*b > 4.0)
			break;
	}
	*(gpu_t + 400*blockIdx.x + blockIdx.y) = i+1;
}
void idlefunc(void)
{
	// DO SOME STUFF
}

void displayfunc(void)
{
	glClear(GL_COLOR_BUFFER_BIT);
	int x;
	int y;

	int *gpu_t;
	hipMalloc((void**)&gpu_t, HEIGHT*WIDTH*sizeof(int));
	printf("Declared array\n");
	int cpu_t[HEIGHT][WIDTH];
	dim3 grid(WIDTH, HEIGHT);
	mandel<<<grid, 1>>>(gpu_t);
	printf("Called the kernel\n");
	hipMemcpy(cpu_t, gpu_t, HEIGHT*WIDTH*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(gpu_t);
	printf("Copied the data back\n");
	for(x=0;x<WIDTH;x++)
	{
		for(y=0;y<HEIGHT;y++)
		{
			glColor3f( 0 , cpu_t[x][y]*1.0/tmax , 0 ) ; // brown
         	//
         	glBegin(GL_POINTS);
         	glVertex2f(x,y);
         	glEnd();
		}
		glutSwapBuffers();
	}
	printf("Displayed data\n");

}

void reshapefunc(int wscr,int hscr)
{
	HEIGHT = hscr;
	WIDTH = wscr;
	glViewport(0,0,(GLsizei)WIDTH,(GLsizei)HEIGHT);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0,1.0*WIDTH,0.0,1.0*HEIGHT);
	glMatrixMode(GL_MODELVIEW);
}
void mousefunc(int button,int state,int xscr,int yscr)
{
   if(button==GLUT_LEFT_BUTTON && state==GLUT_DOWN)
   {
      printf("Left (%d, %d)\n",xscr,HEIGHT-yscr);
   }
   else if(button==GLUT_RIGHT_BUTTON && state==GLUT_DOWN)
   {
      printf("Right (%d, %d)\n",xscr,HEIGHT-yscr);
   }
}
void motionfunc(int xscr,int yscr)
{
   // SHOW MOUSE MOTION
}
void keyfunc(unsigned char key,int xscr,int yscr)
{
	if(key == 32)
	{
		tmax *= 2;
		printf("%d\n", tmax);
	}
}
void specialfunc(int key,int xscr,int yscr)
{
   // ARROW KEYS
}
void closefunc(void)
{
   printf("Window closed.\n");
}

int main(int argc, char *argv[])
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowSize(WIDTH, HEIGHT);
	glutInitWindowPosition(100,50);
	glutCreateWindow("Mandelbrot Set");
	glClearColor(1.0, 1.0, 1.0, 0.0);
	glShadeModel(GL_SMOOTH);

	glutIdleFunc(idlefunc);
	glutDisplayFunc(displayfunc);
	glutReshapeFunc(reshapefunc);
	glutMouseFunc(mousefunc);
	glutMotionFunc(motionfunc);
	glutKeyboardFunc(keyfunc);
	glutSpecialFunc(specialfunc);

	glutMainLoop();

	return 0;
}


// int mandel(double x, double y, int tmax)
// {
// 	double a = 0.0;
// 	double b = 0.0;
// 	int i;
// 	for(i=0; i<tmax; i++)
// 	{
// 		double anew = a*a - b*b + x;
// 		double bnew = 2*a*b + y;
// 		a = anew;
// 		b = bnew;
// 		if(a*a + b*b > 4.0)
// 			break;
// 	}
// 	return (i+1);
// }

void translate(int x, int y, double* a, double* b)
{
	*a = x/100.0 - 2.0;
	*b = y/100.0 - 1.5;
}
