#include "hip/hip_runtime.h"
#include <stdio.h>
#include <GL/glut.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#define WIDTH 800
#define HEIGHT 600


int tmax = 100;
int *gpu_t;
int cpu_t[WIDTH*HEIGHT];
double center_x;
double center_y;
double center_dy;
double center_dx;

// int mandel(float, float, int);
void translate(int, int, double*, double*);
__device__ double translatex(int x, float center_x, float center_dx)
{
	return x*2.0*center_dx/WIDTH + (center_x - center_dx);

}
__device__ double translatey(int y, float center_y, float center_dy)
{
	return y*2.0*center_dy/HEIGHT + (center_y - center_dy);

}

__global__ void mandel(int* gpu_t, int tmax, float center_x, float center_y, float center_dx, float center_dy)
{
	double x = translatex(blockIdx.x, center_x, center_dx);
	double y = translatey(blockIdx.y, center_y, center_dy);
	double a = 0.0;
	double b = 0.0;
	int i;
	for(i=0; i< tmax; i++)
	{
		double anew = a*a - b*b + x;
		double bnew = 2*a*b + y;
		a = anew;
		b = bnew;
		if(a*a + b*b > 4.0)
			break;
	}
	*(gpu_t + blockIdx.x + WIDTH*blockIdx.y) = i+1;
}

void idlefunc(void)
{
	// DO SOME STUFF
}

void displayfunc(void)
{
	if(tmax == 0)
		tmax = 1;
	glClear(GL_COLOR_BUFFER_BIT);
	int x;
	// int y;
	hipMalloc((void**)&gpu_t, HEIGHT*WIDTH*sizeof(int));
	// printf("Declared array\n");
	dim3 grid(WIDTH, HEIGHT);
	mandel<<<grid, 1>>>(gpu_t, tmax, center_x, center_y, center_dx, center_dy);
	// printf("Called the kernel\n");
	hipMemcpy(cpu_t, gpu_t, HEIGHT*WIDTH*sizeof(int), hipMemcpyDeviceToHost);
	// printf("Copied the data back\n");
	for(x=0;x<WIDTH*HEIGHT;x++)
	{
		glColor3f( 0 , cpu_t[x]*1.0/tmax , 0 ) ; // brown
     	//
     	glBegin(GL_POINTS);
     	glVertex2f(x%WIDTH, x/WIDTH);
     	glEnd();
     	// if(x%WIDTH == 0 ) glutSwapBuffers();	
	}
	
	hipFree(gpu_t);
	glutSwapBuffers();
	// exit(0);
	// printf("Displayed data\n");

}

void reshapefunc(int wscr,int hscr)
{
	// HEIGHT = hscr;
	// WIDTH = wscr;
	glViewport(0,0,(GLsizei)WIDTH,(GLsizei)HEIGHT);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0.0,1.0*WIDTH,0.0,1.0*HEIGHT);
	glMatrixMode(GL_MODELVIEW);
}
void mousefunc(int button,int state,int xscr,int yscr)
{
   if(button==GLUT_LEFT_BUTTON && state==GLUT_DOWN)
   {
      // printf("Left (%d, %d)\n",xscr,HEIGHT-yscr);
      double xnew = 0;
      double ynew = 0;
      translate(xscr, HEIGHT-yscr, &xnew, &ynew);
      center_x = xnew;
      center_y = ynew;
      center_dx*= 0.7;
      center_dy*= 0.7;
      printf("(%0.16f, %0.16f. %0.16f, %0.16f)\n", center_x, center_y, center_dx, center_dy);
      glutPostRedisplay();
   }
   else if(button==GLUT_RIGHT_BUTTON && state==GLUT_DOWN)
   {
      
      double xnew = 0;
      double ynew = 0;
      translate(xscr, HEIGHT-yscr, &xnew, &ynew);
      center_x = xnew;
      center_y = ynew;
      center_dx/= 0.7;
      center_dy/= 0.7;
      printf("(%0.16f, %0.16f. %0.16f, %0.16f)\n", center_x, center_y, center_dx, center_dy);
      glutPostRedisplay();
   }
}
void motionfunc(int xscr,int yscr)
{
   // SHOW MOUSE MOTION
}
void keyfunc(unsigned char key,int xscr,int yscr)
{
	if(key == 32)
	{
		tmax *= 2;
		printf("%d\n", tmax);
		glutPostRedisplay();
	}
}
void specialfunc(int key,int xscr,int yscr)
{
	if(key == GLUT_KEY_DOWN)
	{
   		tmax /= 2;
		printf("%d\n", tmax);
		glutPostRedisplay();
	}
	if(key == GLUT_KEY_UP)
	{
   		tmax *= 2;
   		printf("%d\n", tmax);
		glutPostRedisplay();
	}
}
void closefunc(void)
{
   printf("Window closed.\n");
   printf("MEM CLEARED\n");
}

int main(int argc, char *argv[])
{
	center_x = 0.4099716787079002;
	center_y = -0.1482886524476352;
	center_dx = 0.0002682137239328;
	center_dy = 0.0002011602929496;
	tmax = 400;
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowSize(WIDTH, HEIGHT);
	glutInitWindowPosition(100,50);
	glutCreateWindow("Mandelbrot Set");
	glClearColor(1.0, 1.0, 1.0, 0.0);
	glShadeModel(GL_SMOOTH);

	glutIdleFunc(idlefunc);
	glutDisplayFunc(displayfunc);
	glutReshapeFunc(reshapefunc);
	glutMouseFunc(mousefunc);
	glutMotionFunc(motionfunc);
	glutKeyboardFunc(keyfunc);
	glutSpecialFunc(specialfunc);

	glutMainLoop();

	return 0;
}


// int mandel(float x, float y, int tmax)
// {
// 	float a = 0.0;
// 	float b = 0.0;
// 	int i;
// 	for(i=0; i<tmax; i++)
// 	{
// 		float anew = a*a - b*b + x;
// 		float bnew = 2*a*b + y;
// 		a = anew;
// 		b = bnew;
// 		if(a*a + b*b > 4.0)
// 			break;
// 	}
// 	return (i+1);
// }

void translate(int x, int y, double* a, double* b)
{
	*a = x*2.0*center_dx/WIDTH + (center_x - center_dx);
	*b = y*2.0*center_dy/HEIGHT + (center_y - center_dy);
}
